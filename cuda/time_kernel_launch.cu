
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>
#include <string>

// #include "helper_cuda.h"

// #ifdef EXTERN 
// #include "noop.h"
// #else
// static inline void noop() {
//     return;
// }
// #endif

__global__ void noop() {
  return;
}

int main(int argc, char** argv){
    if (argc != 2) {
        fprintf(stderr, "Usage: %s NUMBER_OF_ITERATIONS\n", argv[0]);
        return 1;
    }

    size_t iterations = std::stoll(argv[1]);

    auto start = std::chrono::steady_clock::now();
    for(size_t i=0; i < iterations; ++i) {
      hipError_t err = hipSuccess;
      noop<<<1, 32>>>();
      err = hipGetLastError();

      if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
      }
    }
    auto elapsed = std::chrono::duration_cast<std::chrono::microseconds>(std::chrono::steady_clock::now() - start).count();


    double average_ns = ((double)(elapsed / 1e6) / iterations) * 1e9;

    std::cout << "CUDA kernel launch total [average] runtime for " << iterations
              << " iterations: " << (double)(elapsed / 1e6) << "s ["
              << average_ns << "ns]" << std::endl;

    return 0;
}
